#include "hip/hip_runtime.h"
#include<iostream> 
#include<hip/hip_runtime_api.h> 
#include<thrust/pair.h>
#include<thrust/device_vector.h>
#include<thrust/extrema.h>
#include<chrono> 
#include<cfloat> 

using namespace std; 

int clamp(int x, int a, int b){
    return max(a, min(b,x)); 
}
__device__ int dev_clamp(int x, int a, int b){
    return max(a, min(b,x)); 
}

__global__ void clamp_all(int N, float* d_x, uint8_t* d_y, float y_scale, int y_zeroPoint){
    int i = threadIdx.x + blockIdx.x * blockDim.x; 
    
    if(i < N) d_y[i] = dev_clamp(llrintf(d_x[i]/y_scale)+y_zeroPoint, 0, 255);  
        
}

void dynamic_quantize_linear(int N, float* d_x, uint8_t* d_y){

    float x_min, x_max;
    
    auto start = chrono::high_resolution_clock::now(); 
    thrust::pair<thrust::device_ptr<float>, thrust::device_ptr<float>> tuple;
    tuple = thrust::minmax_element(thrust::device_pointer_cast(d_x), thrust::device_pointer_cast(d_x) + N);
    
    auto stop = chrono::high_resolution_clock::now(); 
    auto duration = chrono::duration_cast<chrono::microseconds>(stop-start); 

    cout << "MinMax time: " << duration.count() << " microseconds" << endl; 
    
    x_min = tuple.first[0];
    x_max = tuple.second[0];

    cout << "minelement " << x_min <<" - maxelement " << x_max << endl;
    
    float y_scale = (x_max-x_min) / (float)(UINT8_MAX-0);
    cout << "y_scale = " << y_scale << endl; 
            
    ///Find zero point
    ///y_zeroPoint = clamp(round((0-x_min)/y_scale), 0, UINT8_MAX); 
    int y_zeroPoint = clamp((int)round((0-x_min)/y_scale), 0, UINT8_MAX); 
                
    cout << "y_zeroPoint = " << y_zeroPoint << endl; 

    unsigned int TB_size = 1024; 
    start = chrono::high_resolution_clock::now(); 

    clamp_all<<<ceil((double)N/(double)TB_size), TB_size>>>(N, d_x, d_y, y_scale, y_zeroPoint);
    stop = chrono::high_resolution_clock::now(); 

    duration = chrono::duration_cast<chrono::microseconds>(stop-start); 

    cout << "Clamp time: " << duration.count() << " microseconds" << endl; 
}



int main(){

    const int N = 10000000;
    
    cout << "For large array (10000000 elements)" << endl; 
         
    float *h_x = (float*) malloc(N*sizeof(float)); 
   
    srand(unsigned(time(nullptr)));
    //generate(x_large.begin(), x_large.end(), (float)rand()/(float)rand());   
    for(int i = 0; i<N; i++){
        h_x[i] = (float)rand()/(float)rand() - (float)rand()/(float)rand(); 
        //cout << x_large[i] << " ";
    }

    float *d_x;

    uint8_t *h_y, *d_y;  
    auto start = chrono::high_resolution_clock::now(); 
     
    hipMalloc(&d_x, N*sizeof(float));
    hipMemcpy(d_x, h_x, N*sizeof(float), hipMemcpyHostToDevice); 
    auto stop = chrono::high_resolution_clock::now(); 

    auto duration = chrono::duration_cast<chrono::microseconds>(stop-start); 

    cout << "Memcpy host to device time: " << duration.count() << " microseconds" << endl; 
    
    h_y = (uint8_t*) malloc(N*sizeof(uint8_t)); 
    hipMalloc(&d_y, N*sizeof(uint8_t));

    dynamic_quantize_linear(N, d_x, d_y);

    start = chrono::high_resolution_clock::now();

    hipMemcpy(h_y, d_y, N*sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stop = chrono::high_resolution_clock::now();

    duration = chrono::duration_cast<chrono::microseconds>(stop-start);    

    cout << "Memcpy device to host time: " << duration.count() << " microseconds" << endl;        


    return 0;
}
